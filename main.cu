#include "hip/hip_runtime.h"
#include <hipDNN.h>

#include <iostream>
#include <fstream>
#include <sstream>
#include <stdio.h>

#include <sys/stat.h>

using std::endl;
using std::cout;

//////////////////////////////////////////////////////////////////////////////
// Error handling
// Adapted from the CUDNN classification code 
// sample: https://developer.nvidia.com/cuDNN

#define FatalError(s) do {                                             \
    std::stringstream _where, _message;                                \
    _where << __FILE__ << ':' << __LINE__;                             \
    _message << std::string(s) + "\n" << __FILE__ << ':' << __LINE__;  \
    std::cerr << _message.str() << "\nAborting...\n";                  \
    hipDeviceReset();                                                 \
    exit(1);                                                           \
} while(0)

#define checkCUDNN(status) do {                                        \
    std::stringstream _error;                                          \
    if (status != HIPDNN_STATUS_SUCCESS) {                              \
      _error << "CUDNN failure: " << hipdnnGetErrorString(status);      \
      FatalError(_error.str());                                        \
    }                                                                  \
} while(0)

#define checkCudaErrors(status) do {                                   \
    std::stringstream _error;                                          \
    if (status != 0) {                                                 \
      _error << "Cuda failure: " << status;                            \
      FatalError(_error.str());                                        \
    }                                                                  \
} while(0)

bool fileExists(const char* file) {
    struct stat buf;
    return (stat(file, &buf) == 0);
}

#define RAW_BOARD_BYTES    (4*3)
#define BOARD_TENSOR_FLOATS (8*8*3)

#include "checkerboard.hpp"

void printBoardTensor(float * boardTensor)
{
    for (int c = 0; c < 3; c++){
        string board_out;
        for (int h = 0; h < 8; h++){
            string row;
            for (int w = 0; w < 8; w++){
                int float_id = c * 8 * 8 + h * 8 + w;
                float ff = boardTensor[float_id];
                string value = " *** ";
                if (ff == 1.0f){
                    value = " 1.0 ";
                }
                row += value;
            }
            row += "\n";
            board_out = row + board_out;
        }
        cout << board_out << endl;
    }
}

__constant__ uint32_t POS_MASK_D[32];

__global__ void raw_game_to_tensor(uint32_t * raw_game, float * game_tensor, size_t num_boards)
{
    size_t board_id = blockIdx.x * (blockDim.x * blockDim.y) + threadIdx.x;     // A board consists of 3 uint32_t bitboards
    if (board_id >= num_boards){
        return;
    }

    size_t bitboard_id = board_id * 3 + threadIdx.y;
    size_t tensor_id_start = bitboard_id * (8 * 8);      // A uint32_t bitboard translates to an 8*8 sparse matrix of floats

    uint32_t board = raw_game[bitboard_id];

    for (size_t bit = 0; bit < 32; bit++){
        size_t fvalue_id;
        size_t fzero_id;
        if ((bit / 4) % 2 == 0){
            fvalue_id = bit * 2;
            fzero_id  = bit * 2 + 1;
            
        } else{
            fvalue_id = bit * 2 + 1;
            fzero_id  = bit * 2;
        }
        float value = 0.0f;
        if (board & POS_MASK_D[bit]){
            value = 1.0f;
        }
        game_tensor[tensor_id_start + fvalue_id] = value;
        game_tensor[tensor_id_start + fzero_id]  = 0.0f;
    }
}

int main()
{
    // READ IN FILE
    std::ifstream infile;

    // Create file name
    static size_t file_num = 1;
    char str[40];
    sprintf(str, "./game_data/games%04lu.bin", file_num++);

    // Check for file.
    if (!fileExists(str)){
        sprintf(str, "./game_data/games%04lu.bin does not exist.", file_num);
        cout << str << endl;
        cout << "Exiting" << endl;
        exit(1);
    }

    // Open and error checking
    infile.open(str, std::ios::in | std::ios::binary);
    if (!infile.is_open()){
        cout << "Failed to open file" << endl;
        cout << "Exiting" << endl;
        exit(1);
    } else{
        cout << str << endl;
    }

    // Get file length
    infile.seekg (0, infile.end);
    int length = infile.tellg();
    infile.seekg (0, infile.beg);

    // Verify valid length
    size_t num_uint = length / sizeof(uint32_t);
    if (length % sizeof(uint32_t) != 0 ||
            num_uint % 3 != 0){
        cout << "Invalid input file size" << endl;
        exit(1);
    }

    // Read in data
    uint32_t raw_game[num_uint];
    infile.read((char *) raw_game, length);

    infile.close();

    // Allocate mem on device
    uint32_t * d_raw_game;
	checkCudaErrors(hipMalloc(&d_raw_game, num_uint * sizeof(uint32_t)));

    // Copy raw game data to device
    checkCudaErrors(hipMemcpy(d_raw_game, raw_game, num_uint * sizeof(uint32_t), hipMemcpyHostToDevice));

    // Allocate mem for game tensor
    int n_boards, c_bitboards, h_rows, w_cols;
    c_bitboards = 3;
    n_boards = num_uint / c_bitboards;
    h_rows = 8;
    w_cols = 8;
    size_t num_game_tensor_floats = num_uint * h_rows * w_cols;
    float * d_game_tensor;
	checkCudaErrors(hipMalloc(&d_game_tensor, num_game_tensor_floats * sizeof(float)));

    // Copy POS_MASK
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(POS_MASK_D), POS_MASK, 32 * sizeof(uint32_t), size_t(0), hipMemcpyHostToDevice));

    // Generate game tensor
    size_t num_blocks = num_uint / 1024 + 1;
    dim3 threadsPerBlock(1024/3, 3);
    raw_game_to_tensor<<<num_blocks,threadsPerBlock>>>(d_raw_game, d_game_tensor, n_boards);
	checkCudaErrors(hipDeviceSynchronize());

    // Copy game tensor data to host
    size_t nb = 3;
    float boardTensor[BOARD_TENSOR_FLOATS * nb];
    checkCudaErrors(hipMemcpy(boardTensor, d_game_tensor, nb * BOARD_TENSOR_FLOATS * sizeof(float), hipMemcpyDeviceToHost));

    for(int i = 0; i < nb; i++){
        printBoardTensor(boardTensor + i * BOARD_TENSOR_FLOATS);
    }

    // Free memory
	checkCudaErrors(hipFree(d_raw_game));
	checkCudaErrors(hipFree(d_game_tensor));

    exit(0);

    size_t version = hipdnnGetVersion();
    if(version/1000 != 4){
        cout << "Not cuDNN v4" << endl;
        cout << "version: " << version << endl;
    }

    hipdnnHandle_t handle;
    checkCUDNN(hipdnnCreate(&handle));

    hipdnnTensorDescriptor_t tensor;

    checkCUDNN(hipdnnCreateTensorDescriptor(&tensor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(tensor,
                                            HIPDNN_TENSOR_NCHW,
                                            HIPDNN_DATA_FLOAT,
                                            n_boards,
                                            c_bitboards,
                                            h_rows,
                                            w_cols));


    
    checkCUDNN(hipdnnDestroyTensorDescriptor(tensor));
    checkCUDNN(hipdnnDestroy(handle));

    return 0;
}
/*
Notes:

Functions:
    hipdnnGetVersion()
    hipdnnCreate(hipdnnHandle_t *)
    hipdnnDestroy(hipdnnHandle_t)

    hipdnnCreateTensorDescriptor(hipdnnTensorDescriptor_t *)
    hipdnnSetTensor4dDescriptor(hipdnnTensorDescriptor_t,
                               hipdnnTensorFormat_t       HIPDNN_TENSOR_NCHW,
                               hipdnnDataType_t           HIPDNN_DATA_FLOAT,
                               int                       n_boards,
                               int                       c_bitboards,
                               int                       h_rows,
                               int                       w_cols)
    

*/
