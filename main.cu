#include "hip/hip_runtime.h"
#include <hipDNN.h>/*{{{*/

#include <iostream>
#include <fstream>
#include <sstream>
#include <stdio.h>

#include <sys/stat.h>

using std::endl;
using std::cout;/*}}}*/

// Error handling/*{{{*/
// Adapted from the CUDNN classification code 
// sample: https://developer.nvidia.com/cuDNN

#define FatalError(s) do {                                             \
    std::stringstream _where, _message;                                \
    _where << __FILE__ << ':' << __LINE__;                             \
    _message << std::string(s) + "\n" << __FILE__ << ':' << __LINE__;  \
    std::cerr << _message.str() << "\nAborting...\n";                  \
    hipDeviceReset();                                                 \
    exit(1);                                                           \
} while(0)

#define checkCUDNN(status) do {                                        \
    std::stringstream _error;                                          \
    if (status != HIPDNN_STATUS_SUCCESS) {                              \
      _error << "CUDNN failure: " << hipdnnGetErrorString(status);      \
      FatalError(_error.str());                                        \
    }                                                                  \
} while(0)

#define checkCudaErrors(status) do {                                   \
    std::stringstream _error;                                          \
    if (status != 0) {                                                 \
      _error << "Cuda failure: " << status;                            \
      FatalError(_error.str());                                        \
    }                                                                  \
} while(0)/*}}}*/

bool fileExists(const char* file) {/*{{{*/
    struct stat buf;
    return (stat(file, &buf) == 0);
}/*}}}*/

#define RAW_BOARD_BYTES    (4*3)
#define BOARD_TENSOR_FLOATS (8*8*3)

#include "checkerboard.hpp"

void printBoardTensor(float * boardTensor)/*{{{*/
{
    for (int c = 0; c < 3; c++){
        string board_out;
        for (int h = 0; h < 8; h++){
            string row;
            for (int w = 0; w < 8; w++){
                int float_id = c * 8 * 8 + h * 8 + w;
                float ff = boardTensor[float_id];
                string value = " *** ";
                if (ff == 1.0f){
                    value = " 1.0 ";
                }
                row += value;
            }
            row += "\n";
            board_out = row + board_out;
        }
        cout << board_out << endl;
    }
}/*}}}*/

__constant__ uint32_t POS_MASK_D[32];

#pragma pack(push, 1)
struct GameStat/*{{{*/
{
    bool win = false;
    uint16_t num_moves = 0;
};/*}}}*/
#pragma pack(pop)

__global__ void gen_180_raw_game(uint32_t * raw_game, uint32_t * raw_game_180, size_t num_boards)/*{{{*/
{
    size_t board_id = blockIdx.x * (blockDim.x * blockDim.y) + threadIdx.x;     // A board consists of 3 uint32_t bitboards
    if (board_id >= num_boards){
        return;
    }

    size_t bitboard_id = board_id * 3 + threadIdx.y;

    size_t bitboard180_id;
    if(threadIdx.y == 0){
        bitboard180_id = board_id * 3 + 1;
    } else if (threadIdx.y == 1){
        bitboard180_id = board_id * 3;
    } else{
        bitboard180_id = board_id * 3 + 2;
    }

    uint32_t bitboard = raw_game[bitboard_id];
    uint32_t bitboard180 = 0;

    for (int i = 0; i < 32; i++){
        if (bitboard & POS_MASK_D[i]){
            bitboard180 |= POS_MASK_D[31 - i];
        }
    }

    raw_game_180[bitboard180_id] = bitboard180;
}/*}}}*/

__global__ void gen_label_tensor(float * label_tensor, bool win)/*{{{*/
{
    size_t num_moves = blockDim.x;
    size_t game_id = threadIdx.x * 2;

    float value = ((float)(threadIdx.x)) / (num_moves - 1);
    if (!win){
        value *= -1.0f;
    }

    label_tensor[game_id] = value;
    label_tensor[game_id + 1] = -1.0f * value;
}/*}}}*/

__global__ void raw_game_to_tensor(uint32_t * raw_game, uint32_t * raw_game_180, float * game_tensor, size_t num_boards)/*{{{*/
{
    size_t board_id = blockIdx.x * (blockDim.x * blockDim.y) + threadIdx.x;     // A board consists of 3 uint32_t bitboards
    if (board_id >= num_boards){
        return;
    }

    size_t bitboard_id = board_id * 3 + threadIdx.y;
    size_t tensor_id_start = ((board_id * 2 + threadIdx.z) * 3 + threadIdx.y) * (8 * 8);      // A uint32_t bitboard translates to an 8*8 sparse matrix of floats

    uint32_t board;
    if (threadIdx.z == 0){
        board = raw_game[bitboard_id];
    } else{
        board = raw_game_180[bitboard_id];
    }

    for (size_t bit = 0; bit < 32; bit++){
        size_t fvalue_id;
        size_t fzero_id;
        if ((bit / 4) % 2 == 0){
            fvalue_id = bit * 2;
            fzero_id  = bit * 2 + 1;
            
        } else{
            fvalue_id = bit * 2 + 1;
            fzero_id  = bit * 2;
        }
        float value = 0.0f;
        if (board & POS_MASK_D[bit]){
            value = 1.0f;
        }
        game_tensor[tensor_id_start + fvalue_id] = value;
        game_tensor[tensor_id_start + fzero_id]  = 0.0f;
    }
}/*}}}*/

int main()/*{{{*/
{
    // READ IN FILE/*{{{*//*{{{*/
    std::ifstream binfile;
    std::ifstream infofile;

    // Create file name
    static size_t file_num = 2;
    char str1[40];
    char str2[40];
    sprintf(str1, "./game_data/games%04lu.bin", file_num);
    sprintf(str2, "./game_data/games%04lu.info", file_num);

    // Check for file.
    if (!fileExists(str1) || !fileExists(str2)){
        sprintf(str1, "./game_data/games%04lu.bin does not exist.", file_num);
        sprintf(str2, " or ./game_data/games%04lu.info does not exist.", file_num);
        cout << str1 << endl;
        cout << str2 << endl;
        cout << "Exiting" << endl;
        exit(1);
    }

    // Open and error checking
    binfile.open(str1, std::ios::in | std::ios::binary);
    infofile.open(str2, std::ios::in | std::ios::binary);
    if (!binfile.is_open() || !infofile.is_open()){
        cout << "Failed to open file" << endl;
        cout << "Exiting" << endl;
        exit(1);
    } else{
        cout << str1 << endl;
        cout << str2 << endl;
    }

    // Get file binlength
    binfile.seekg (0, binfile.end);
    int binlength = binfile.tellg();
    binfile.seekg (0, binfile.beg);

    infofile.seekg (0, infofile.end);
    int infolength = infofile.tellg();
    infofile.seekg (0, infofile.beg);

    // Verify valid binlength
    size_t num_uint = binlength / sizeof(uint32_t);
    if (binlength % sizeof(uint32_t) != 0 ||
            num_uint % 3 != 0){
        cout << "Invalid bin file size" << endl;
        exit(1);
    }

    if (infolength / sizeof(GameStat) != 100){
        cout << infolength << endl;
        cout << "Invalid info file size" << endl;
        exit(1);
    }
    /*}}}*/

    // Read in data
    uint32_t raw_game[num_uint];
    binfile.read((char *) raw_game, binlength);
    binfile.close();

    GameStat gstat[100];
    infofile.read((char *) gstat, infolength);
    infofile.close();
    /*}}}*/

    // Allocate mem on device/*{{{*/
    uint32_t * d_raw_game;
    uint32_t * d_180_raw_game;
	checkCudaErrors(hipMalloc(&d_raw_game, num_uint * sizeof(uint32_t)));
	checkCudaErrors(hipMalloc(&d_180_raw_game, num_uint * sizeof(uint32_t)));
    /*}}}*/

    // Copy raw game data to device/*{{{*/
    checkCudaErrors(hipMemcpy(d_raw_game, raw_game, num_uint * sizeof(uint32_t), hipMemcpyHostToDevice));
    /*}}}*/

    // Copy pos_mask/*{{{*/
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(POS_MASK_D), POS_MASK, 32 * sizeof(uint32_t), size_t(0), hipMemcpyHostToDevice));
    /*}}}*/

    // Generate 180 games/*{{{*/
    size_t num_blocks = num_uint / 1024 + 1;
    dim3 threadsPerBlock(1024/3, 3);
    gen_180_raw_game<<<num_blocks, threadsPerBlock>>>(d_raw_game, d_180_raw_game, num_uint / 3);
	checkCudaErrors(hipDeviceSynchronize());
    /*}}}*/

    // Allocate mem for game tensor and label tensor/*{{{*/
    int n_boards, c_bitboards, h_rows, w_cols;
    c_bitboards = 3;
    n_boards = num_uint / c_bitboards;
    h_rows = 8;
    w_cols = 8;
    size_t num_game_tensor_floats = num_uint * h_rows * w_cols;
    float * d_game_tensor;
    float * d_label_tensor;
	checkCudaErrors(hipMalloc(&d_game_tensor, 2 * num_game_tensor_floats * sizeof(float)));
	checkCudaErrors(hipMalloc(&d_label_tensor, 2 * n_boards * sizeof(float)));
    /*}}}*/

    // generate game and label tensor/*{{{*/
    num_blocks = num_uint / 1024 + 1;
    threadsPerBlock = dim3(1024/6, 3, 2);
    raw_game_to_tensor<<<num_blocks,threadsPerBlock>>>(d_raw_game, d_180_raw_game, d_game_tensor, n_boards);

    size_t num_moves = 0;
    for (size_t i = 0; i < 100; i++){
        float * start_label = d_label_tensor + 2 * num_moves;
        num_moves += gstat[i].num_moves;
        gen_label_tensor<<<1,gstat[i].num_moves>>>(start_label, gstat[i].win);
    }
	checkCudaErrors(hipDeviceSynchronize());
    /*}}}*/

    // Copy game tensor data to host/*{{{*/
    size_t nb = gstat[0].num_moves * 2;
    float boardTensor[BOARD_TENSOR_FLOATS * nb];
    float labels[nb];
    checkCudaErrors(hipMemcpy(boardTensor, d_game_tensor, nb * BOARD_TENSOR_FLOATS * sizeof(float), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(labels, d_label_tensor, nb * sizeof(float), hipMemcpyDeviceToHost));
    /*}}}*/

    // Print out tensor board/*{{{*/
    for(int i = 0; i < nb; i++){
        printBoardTensor(boardTensor + i * BOARD_TENSOR_FLOATS);
        float f = labels[i];
        cout << i/2 << ": " << f << endl << endl;
    }
    /*}}}*/

    // Free memory/*{{{*/
	checkCudaErrors(hipFree(d_raw_game));
	checkCudaErrors(hipFree(d_180_raw_game));
	checkCudaErrors(hipFree(d_game_tensor));
	checkCudaErrors(hipFree(d_label_tensor));
    /*}}}*/

    exit(0);

    // CUDNN SCRATCH/*{{{*/
    size_t version = hipdnnGetVersion();
    if(version/1000 != 4){
        cout << "Not cuDNN v4" << endl;
        cout << "version: " << version << endl;
    }

    hipdnnHandle_t handle;
    checkCUDNN(hipdnnCreate(&handle));

    hipdnnTensorDescriptor_t tensor;

    checkCUDNN(hipdnnCreateTensorDescriptor(&tensor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(tensor,
                                            HIPDNN_TENSOR_NCHW,
                                            HIPDNN_DATA_FLOAT,
                                            n_boards,
                                            c_bitboards,
                                            h_rows,
                                            w_cols));


    
    checkCUDNN(hipdnnDestroyTensorDescriptor(tensor));
    checkCUDNN(hipdnnDestroy(handle));
    /*}}}*/

    return 0;
}/*}}}*/

// Notes:/*{{{*/
/*
Functions:
    hipdnnGetVersion()
    hipdnnCreate(hipdnnHandle_t *)
    hipdnnDestroy(hipdnnHandle_t)

    hipdnnCreateTensorDescriptor(hipdnnTensorDescriptor_t *)
    hipdnnSetTensor4dDescriptor(hipdnnTensorDescriptor_t,
                               hipdnnTensorFormat_t       HIPDNN_TENSOR_NCHW,
                               hipdnnDataType_t           HIPDNN_DATA_FLOAT,
                               int                       n_boards,
                               int                       c_bitboards,
                               int                       h_rows,
                               int                       w_cols)
    
*/
/*}}}*/
