#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <sstream>
//#include <hiprand.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <chrono>
#include <ctime>
#include "checkerboard_gpu.hpp"

using std::endl;
using std::cout;

// Error handling/*{{{*/
// Adapted from the CUDNN classification code 
// sample: https://developer.nvidia.com/cuDNN

#define FatalError(s) do {                                             \
    std::stringstream _where, _message;                                \
    _where << __FILE__ << ':' << __LINE__;                             \
    _message << std::string(s) + "\n" << __FILE__ << ':' << __LINE__;  \
    std::cerr << _message.str() << "\nAborting...\n";                  \
    hipDeviceReset();                                                 \
    exit(1);                                                           \
} while(0)

#define checkCUDNN(status) do {                                        \
    std::stringstream _error;                                          \
    if (status != HIPDNN_STATUS_SUCCESS) {                              \
      _error << "CUDNN failure: " << hipdnnGetErrorString(status);      \
      FatalError(_error.str());                                        \
    }                                                                  \
} while(0)

#define checkCudaErrors(status) do {                                   \
    std::stringstream _error;                                          \
    if (status != 0) {                                                 \
      _error << "Cuda failure: " << status;                            \
      FatalError(_error.str());                                        \
    }                                                                  \
} while(0)/*}}}*/

using std::chrono::system_clock;

__constant__ uint32_t POS_MASK_D[32];

__global__ void setup_kernel(hiprandState *state, unsigned long long r_offset)
{
    unsigned long long idx = threadIdx.x + blockDim.x * (blockIdx.x * gridDim.y + blockIdx.y);
    unsigned long long sequence = threadIdx.x;
    unsigned long long seed = (idx + 1) * r_offset;
    hiprand_init(seed, sequence, r_offset, &state[idx]);
}

#define MAX_MOVES 200
__global__ void random_descent( hiprandState * state, 
        BitBoard_gpu * d_bb, unsigned long long * d_wins)
{
    __shared__ unsigned long long wins;
    if (threadIdx.x == 0){
        wins = 0;
    }
    __syncthreads();

    int idx = threadIdx.x + blockDim.x * (blockIdx.x * gridDim.y + blockIdx.y);
    hiprandState localState = state[idx];

    BitBoard_gpu bb = *(d_bb + blockIdx.x);
    bool player = bb.turn;  // SWITCH BACK TO !

    auto children = gen_children_gpu(bb);
    size_t n_moves = 0;
    size_t max_b = 0;
    float frand;
    while(children.size && n_moves < MAX_MOVES){
        n_moves++;
        size_t b = children.size;
        if (b > max_b){
            max_b = b;
        }
        frand = hiprand_uniform(&localState);
        int irand = frand * b;

        bb = children.bb_ary[irand];
        delete [] children.bb_ary;
        children = gen_children_gpu(bb);
    }

    bool winner = bb.turn;
    if (children.size > 0){
        delete [] children.bb_ary;
        winner = !player;
        if (true){
            size_t red_count = bit_count_gpu(bb.red_pos);
            size_t red_king_count = bit_count_gpu(bb.red_pos & bb.king_pos);
            size_t red_score = red_count + red_king_count;

            size_t blk_count = bit_count_gpu(bb.blk_pos);
            size_t blk_king_count = bit_count_gpu(bb.blk_pos & bb.king_pos);
            size_t blk_score = blk_count + blk_king_count;

            if (player == RED){
                if (red_score > blk_score){
                    winner = player;
                }
            } else{
                if (blk_score > red_score){
                    winner = player;
                }
            }
        }
    }
            
    if (winner == player && n_moves < MAX_MOVES){
        //printf("%llu moves\n%llu max\nWinner\n", n_moves, max_b);
        atomicAdd(&wins, 1);
    } //else{
        //printf("%llu moves\n%llu max\nLoser\n", n_moves, max_b);
    //}

    state[idx] = localState;
    __syncthreads();
    if (threadIdx.x == 0){
        atomicAdd(d_wins + blockIdx.x, wins);
    }
}

int main(int argc, char *argv[])
{
    int num_repeat = 15;
    int num_plays = 1024;

    if(argc == 2){
        num_repeat = atoi(argv[1]);
    }
    printf("%d repeat\n", num_repeat);

    BitBoard board;
    BitBoard_gpu * d_board;
    auto children = gen_children(board);
    int size = children.size();
    hiprandState * d_state;
    checkCudaErrors(hipMalloc((void **)&d_state, size * num_repeat * num_plays * sizeof(hiprandState)));
    unsigned long long time = system_clock::to_time_t(system_clock::now());
    checkCudaErrors(hipDeviceSynchronize());
    dim3 blocks(size,num_repeat);
    setup_kernel<<<blocks,num_plays>>>(d_state, time);

    checkCudaErrors(hipMalloc(&d_board, size * sizeof(hiprandState)));
    checkCudaErrors(hipMemcpy(d_board, &children[0], size * sizeof(BitBoard), hipMemcpyHostToDevice));

    unsigned long long * d_wins;
    checkCudaErrors(hipMalloc(&d_wins, size * sizeof(unsigned long)));
    checkCudaErrors(hipMemset(d_wins, 0, size * sizeof(unsigned long)));

    checkCudaErrors(hipDeviceSynchronize());
    cout << "Calling random_descent" << endl;
    for (int i = 0; i < 10; i++){
        cout << "i: " << i << endl;
        random_descent<<<blocks,num_plays>>>(d_state, d_board, d_wins);
        checkCudaErrors(hipDeviceSynchronize());
    }

    unsigned long long wins[size];
    checkCudaErrors(hipMemcpy(wins, d_wins, size * sizeof(unsigned long long), hipMemcpyDeviceToHost));
    checkCudaErrors(hipDeviceSynchronize());

    for (auto w : wins){
        cout << w << endl;
        cout << (double)w/(num_plays*num_repeat*10) << endl;
    }

    hipFree(d_state);
    hipFree(d_wins);
    hipFree(d_board);
}
